#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include "device.h"

extern "C"
cuda_stream_handle create_cuda_stream() {
    hipStream_t retval;
    hipStreamCreate(&retval);
    return static_cast<cuda_stream_handle>(retval);
}

extern "C"
cuda_event_handle create_cuda_event() {
    hipEvent_t stop;
    hipEventCreate(&stop);
    return static_cast<cuda_event_handle>(stop);
}

extern "C" 
void cuda_stream_event_record(cuda_stream_handle strm_hdl, cuda_event_handle eve_hdl){
    hipEventRecord(static_cast<hipEvent_t>(eve_hdl), static_cast<hipStream_t>(strm_hdl));
}

extern "C" 
int cuda_check(cuda_event_handle eve_hdl){
    if (hipEventQuery(static_cast<hipEvent_t>(eve_hdl))==hipErrorNotReady)
       return 0;
    else 
       return 1;
}
